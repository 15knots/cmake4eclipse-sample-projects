#include "hip/hip_runtime.h"
/*
** Hello World using CUDA
**
** The string "Hello World!" is mangled then restored using a common CUDA idiom
**
** Byron Galbraith
** 2009-02-18
*/
#include <hip/hip_runtime.h>
#include <stdio.h>

// Clicking 'Open Declaration' should open config.h
#include "config.h"

// Device kernel
__global__ void
helloWorld(char* str)
{
    // determine where in the thread grid we are
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // unmangle output
    str[idx] += idx;
}

// Host function
int main(int argc, char** argv)
{
#if MACRO_FROM_C_INCLPATH_PRJ-0
// this should not show up with grey background in the editor
	puts("Hello World!" );
#endif
#if MACRO_FROM_COMMANDLINE-0
	// this should not show up with grey background in the editor
	puts("Hello, hello World!" );
#endif

	int i;
    // desired output
    char str[] = "Hello World!";
    // mangle contents of output
    // the null character is left intact for simplicity
    for(i = 0; i < 12; i++)
        str[i] -= i;
    // allocate memory on the device
    char *d_str;
    size_t size = sizeof(str);
    hipMalloc((void**)&d_str, size);
    // copy the string to the device
    hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);
    // set the grid and block sizes
    dim3 dimGrid(2);   // one block per word
    dim3 dimBlock(6); // one thread per character
    // invoke the kernel
    helloWorld<<< dimGrid, dimBlock >>>(d_str);
    // retrieve the results from the device
    hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);
    // free up the allocated memory on the device
    hipFree(d_str);
    // everyone's favorite part
    printf("%s\n", str);
    return 0;
}
